#include "hip/hip_runtime.h"
/**
 * @file test_tile_load_store_v4.cu
 * @author     Adam Rogowiec
 *
 * This file is an integral part of the master thesis entitled:
 * "Elaboration and implementation in CUDA technology parallel version of
 * estimation of multidimensional random variable density function ridge
 * detection algorithm."
 * , which is conducted under the supervision of prof. dr hab. inż. Marek
 * Nałęcz.
 *
 * Institute of Control and Computation Engineering Faculty of Electronics and
 * Information Technology Warsaw University of Technology 2016
 */
#define HIPCUB_STDERR
#define BLOCK_TILE_LOAD_V4 1

#include "rd/utils/cmd_line_parser.hpp"
#include "rd/utils/utilities.hpp"
#include "rd/utils/memory.h" 
#include "rd/utils/rd_params.hpp"
#include "rd/utils/name_traits.hpp"
#include "rd/utils/graph_drawer.hpp"
#include "rd/gpu/util/dev_samples_set.cuh"
 
#include "rd/gpu/agent/agent_memcpy.cuh" 
#include "rd/gpu/device/samples_generator.cuh"

#include "cub/test_util.h"
#include "cub/util_ptx.cuh"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <string>
#include <vector>
#include <sstream>
#include <fstream>
#include <iomanip>
#include <typeinfo>
#include <stdexcept>
#include <utility>

//------------------------------------------------------------
//  GLOBAL CONSTANTS / VARIABLES
//------------------------------------------------------------

static const std::string LOG_FILE_NAME_SUFFIX = "_io-timings_v4.txt";

std::ofstream * g_logFile       = nullptr;
bool            g_drawAllGraphs = false;
bool            g_drawGraphs    = false;
std::string     g_devName;

std::vector<std::vector<float>> g_bestPerf;
const float                     g_graphColStep = 0.3f;
const int                       g_graphNCol = 8;        // group columns count 
const int                       g_graphNGroups = 6;     // number of dimensions to plot
    
#if defined(RD_DEBUG) || defined(RD_PROFILE)
const int g_iterations = 1;
#else
const int g_iterations = 100;
#endif

/******************************************************************************
 * Load Store kernel entry point
 *****************************************************************************/

template<
    typename                    BlockTileLoadPolicyT,
    typename                    BlockTileStorePolicyT,
    int                         DIM,
    rd::DataMemoryLayout        MEM_LAYOUT,
    rd::DataMemoryLayout        PRIVATE_MEM_LAYOUT,
    rd::gpu::BlockTileIOBackend IO_BACKEND,
    typename                    SampleT,
    typename                    OffsetT>
__launch_bounds__ (int(BlockTileLoadPolicyT::BLOCK_THREADS))
__global__ void deviceTileProcessingKernel(
    SampleT const *         d_in,
    SampleT *               d_out,
    int                     numPoints,
    OffsetT                 offset,
    OffsetT                 stride)
{
    typedef rd::gpu::AgentMemcpy<
        BlockTileLoadPolicyT,
        BlockTileStorePolicyT,
        DIM,
        MEM_LAYOUT,
        PRIVATE_MEM_LAYOUT,
        IO_BACKEND,
        OffsetT,
        SampleT> AgentMemcpyT;

    AgentMemcpyT(d_in, d_out).copyRange(offset, numPoints, stride);
    // AgentMemcpyT(d_in, d_out).copyRange(offset, numPoints, stride, true);
}

struct KernelConfig
{
    int blockThreads;
    int itemsPerThread;
};

//---------------------------------------------------------------------
// Kernel Invocation
//---------------------------------------------------------------------

template <
    typename    SampleT,
    typename    OffsetT, 
    typename    LoadStoreKernelPtrT>
static hipError_t invoke(
    SampleT const *         d_in,
    SampleT *               d_out,
    int                     numPoints,
    OffsetT                 offset,
    OffsetT                 stride,
    hipStream_t            stream,
    bool                    debugSynchronous,
    LoadStoreKernelPtrT     kernelPtr,
    KernelConfig            kernelConfig)
{
    hipError_t error = hipSuccess;
    do
    {           
        // Get device ordinal
        int deviceOrdinal;
        if (HipcubDebug(error = hipGetDevice(&deviceOrdinal))) break;

        // Get SM count
        int smCount;
        if (HipcubDebug(error = hipDeviceGetAttribute(&smCount, hipDeviceAttributeMultiprocessorCount, deviceOrdinal))) break;

        // get SM occupancy
        int smOccupancy;
        if (HipcubDebug(error = hipcub::MaxSmOccupancy(
            smOccupancy,
            kernelPtr,
            kernelConfig.blockThreads)
        )) break;

        dim3 loadStoreGridSize(1);
        loadStoreGridSize.x = smCount * smOccupancy * 4;

        if (debugSynchronous)
        {
            printf("Invoking deviceLoadStoreKernel<<<%d, %d, 0, %lld>>> numPoints: %d, itemsPerThread: %d, offset %d\n",
                loadStoreGridSize.x, kernelConfig.blockThreads, (long long)stream, numPoints, kernelConfig.itemsPerThread, offset);
        }

        kernelPtr<<<loadStoreGridSize.x, kernelConfig.blockThreads, 0, stream>>>(
            d_in,
            d_out,
            numPoints,
            offset,
            stride);

        // Check for failure to launch
        if (HipcubDebug(error = hipPeekAtLastError())) break;
        // Sync the stream if specified to flush runtime errors
        if (debugSynchronous && (HipcubDebug(error = hipcub::SyncStream(stream)))) break;

    } while (0);

    return error;
}

//------------------------------------------------------------
//  KERNEL DISPATCH
//------------------------------------------------------------

template <
    int                         BLOCK_THREADS,
    int                         POINTS_PER_THREAD,
    int                         DIM,
    hipcub::CacheLoadModifier      LOAD_MODIFIER,
    hipcub::CacheStoreModifier     STORE_MODIFIER,
    rd::DataMemoryLayout        MEM_LAYOUT,
    rd::DataMemoryLayout        PRIVATE_MEM_LAYOUT,
    rd::gpu::BlockTileIOBackend IO_BACKEND,
    typename                    OffsetT,                     
    typename                    T>
void dispatchTileProcessingKernel(
    T const *           d_in,
    T *                 d_out,
    int                 numPoints,
    OffsetT             offset,
    OffsetT             stride,
    int                 iterations,
    bool                debugSynchronous = false)
{
    typedef rd::gpu::BlockTileLoadPolicy<
        BLOCK_THREADS,
        POINTS_PER_THREAD,
        LOAD_MODIFIER> BlockTileLoadPolicyT;

    typedef rd::gpu::BlockTileStorePolicy<
        BLOCK_THREADS,
        POINTS_PER_THREAD,
        STORE_MODIFIER> BlockTileStorePolicyT;

    KernelConfig tileProcessingConfig;
    tileProcessingConfig.blockThreads = BLOCK_THREADS;
    tileProcessingConfig.itemsPerThread = POINTS_PER_THREAD;

    for (int i = 0; i < iterations; ++i)
    {
        CubDebugExit(invoke(
            d_in,
            d_out,
            numPoints,
            offset,
            stride,
            0,
            debugSynchronous,
            deviceTileProcessingKernel<BlockTileLoadPolicyT, BlockTileStorePolicyT, DIM, MEM_LAYOUT, PRIVATE_MEM_LAYOUT, IO_BACKEND, T, OffsetT>,
            tileProcessingConfig));
    }
}

//------------------------------------------------------------
//  Benchmark helper structures
//------------------------------------------------------------

struct KernelParametersConf
{
    int                         BLOCK_THREADS;
    int                         POINTS_PER_THREAD;
    int                         DIM;
    hipcub::CacheLoadModifier      LOAD_MODIFIER;
    hipcub::CacheStoreModifier     STORE_MODIFIER;
    rd::DataMemoryLayout        MEM_LAYOUT;
    rd::DataMemoryLayout        PRIVATE_MEM_LAYOUT;
    rd::gpu::BlockTileIOBackend IO_BACKEND;
    float                       avgMillis;
    float                       gigaBandwidth;

    KernelParametersConf()
    :
        LOAD_MODIFIER(hipcub::LOAD_DEFAULT),
        STORE_MODIFIER(hipcub::STORE_DEFAULT),
        MEM_LAYOUT(rd::ROW_MAJOR),
        PRIVATE_MEM_LAYOUT(rd::ROW_MAJOR),
        IO_BACKEND(rd::gpu::IO_BACKEND_CUB)
    {}

    KernelParametersConf(
        int                         _DIM,
        hipcub::CacheLoadModifier      _LOAD_MODIFIER,
        hipcub::CacheStoreModifier     _STORE_MODIFIER,
        rd::DataMemoryLayout        _MEM_LAYOUT,
        rd::DataMemoryLayout        _PRIVATE_MEM_LAYOUT,
        rd::gpu::BlockTileIOBackend _IO_BACKEND)
    :
        DIM(_DIM),
        LOAD_MODIFIER(_LOAD_MODIFIER),
        STORE_MODIFIER(_STORE_MODIFIER),
        MEM_LAYOUT(_MEM_LAYOUT),
        PRIVATE_MEM_LAYOUT(_PRIVATE_MEM_LAYOUT),
        IO_BACKEND(_IO_BACKEND)
    {}

};

typedef std::pair<float, float> KernelPerfT;

//------------------------------------------------------------
//  TEST CONFIGURATION AND RUN
//------------------------------------------------------------

template <
    int                         BLOCK_THREADS,
    int                         POINTS_PER_THREAD,
    int                         DIM,
    hipcub::CacheLoadModifier      LOAD_MODIFIER,
    hipcub::CacheStoreModifier     STORE_MODIFIER,
    rd::DataMemoryLayout        MEM_LAYOUT,
    rd::DataMemoryLayout        PRIVATE_MEM_LAYOUT,
    rd::gpu::BlockTileIOBackend IO_BACKEND,
    typename                    OffsetT,                     
    typename                    T>
KernelPerfT runTileProcessing(
    rd::RDParams<T> const &         rdp,
    T const *                       d_in,
    T *                             d_out,
    T const *                       h_in,
    OffsetT                         offset,
    OffsetT                         stride)
{
    std::cout << rd::HLINE << std::endl;
    std::cout << "runTestLoadStore:" << std::endl;
    std::cout << "blockThreads: " << BLOCK_THREADS 
              << ", pointsPerThread: " << POINTS_PER_THREAD
              << ", load modifier: " << rd::LoadModifierNameTraits<LOAD_MODIFIER>::name
              << ", store modifier: " << rd::StoreModifierNameTraits<STORE_MODIFIER>::name
              << ", mem layout: " << rd::DataMemoryLayoutNameTraits<MEM_LAYOUT>::name
              << ", priv mem layout: " << rd::DataMemoryLayoutNameTraits<PRIVATE_MEM_LAYOUT>::name
              << ", io backend: " << rd::BlockTileIONameTraits<IO_BACKEND>::name
              << ", numPoints: " << rdp.np << "\n";
    /*
     *  Allocate output host containers for correctness check
     */
    T * h_out = new T[rdp.np * DIM];

    // Run warm-up/correctness iteration
    dispatchTileProcessingKernel<BLOCK_THREADS, POINTS_PER_THREAD, DIM, LOAD_MODIFIER, STORE_MODIFIER, MEM_LAYOUT, PRIVATE_MEM_LAYOUT, IO_BACKEND>(
        d_in, d_out, rdp.np, offset, stride, 1, true);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(h_out, d_out, rdp.np * DIM * sizeof(T), hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());

    bool result = rd::checkResult(h_in, h_out, rdp.np * DIM);
    if (result)
    {
        std::cout << ">>>> CORRECT!\n";
    }
    else
    {
        std::cout << ">>>> ERROR!" << std::endl;
        // clean-up
        delete[] h_out;
        throw std::logic_error("ERROR! incorrect results!");
    }

    // Measure performance

    GpuTimer timer;
    float elapsedMillis;

    #ifdef RD_PROFILE
    hipProfilerStart();
    #endif
    timer.Start();

    dispatchTileProcessingKernel<BLOCK_THREADS, POINTS_PER_THREAD, DIM, LOAD_MODIFIER, STORE_MODIFIER, MEM_LAYOUT, PRIVATE_MEM_LAYOUT, IO_BACKEND>(
        d_in, d_out, rdp.np, offset, stride, g_iterations);

    timer.Stop();
    elapsedMillis = timer.ElapsedMillis();
    checkCudaErrors(hipDeviceSynchronize());
    #ifdef RD_PROFILE
    hipProfilerStop();
    #endif

    float avgMillis = elapsedMillis / g_iterations;
    float gigaRate = float(rdp.np * DIM) / avgMillis / 1000.0 / 1000.0;
    float gigaBandwidth = gigaRate * 2 * sizeof(T);

    std::cout << avgMillis << " avg ms, "
              // << gigaRate << " billion samples/s, "
              // << gigaRate / DIM << " billion points/s " 
              << gigaBandwidth << " logical GB/s\n";

    if (rdp.verbose)
    {
        *g_logFile << POINTS_PER_THREAD << " " << BLOCK_THREADS << " " << avgMillis << " " << gigaBandwidth << "\n";
    }

    // clean-up
    delete[] h_out;

    return std::make_pair(avgMillis, gigaBandwidth);
}


/*
 *  Specialization for testing different points per thread
 */
template <
    int                         DIM,
    hipcub::CacheLoadModifier      LOAD_MODIFIER,
    hipcub::CacheStoreModifier     STORE_MODIFIER,
    rd::DataMemoryLayout        MEM_LAYOUT,
    rd::DataMemoryLayout        PRIVATE_MEM_LAYOUT,
    rd::gpu::BlockTileIOBackend IO_BACKEND,
    typename                    OffsetT,                     
    typename                    T>
KernelParametersConf testBlockPointsPerThreadConf(
    rd::RDParams<T> const &         rdp,
    T const *                       d_in,
    T *                             d_out,
    T const *                       h_in,
    OffsetT                         offset,
    OffsetT                         stride)
{
    if (rdp.verbose)
    {
        *g_logFile << "%\n testBlockPointsPerThreadConf: "
            << ", load modifier: " << rd::LoadModifierNameTraits<LOAD_MODIFIER>::name
            << ", store modifier: " << rd::StoreModifierNameTraits<STORE_MODIFIER>::name
            << ", mem layout: " << rd::DataMemoryLayoutNameTraits<MEM_LAYOUT>::name
            << ", priv mem layout: " << rd::DataMemoryLayoutNameTraits<PRIVATE_MEM_LAYOUT>::name
            << ", io backend: " << rd::BlockTileIONameTraits<IO_BACKEND>::name
            << ", numPoints: " << rdp.np << "\n";
    }

    KernelParametersConf bestKernelParams(DIM, LOAD_MODIFIER, STORE_MODIFIER, MEM_LAYOUT, PRIVATE_MEM_LAYOUT, IO_BACKEND);
    KernelPerfT bestPerf = std::make_pair(1e10f, -1.0f);


    typedef std::pair<int, std::vector<T>> graphLineDataT;
    std::vector<graphLineDataT> graphData;

    auto processResult = [&](int bs, int ppt, KernelPerfT kp)
    {
        if (kp.second > bestPerf.second)
        {
            bestPerf.first = kp.first;
            bestPerf.second = kp.second;
            bestKernelParams.avgMillis = kp.first;
            bestKernelParams.gigaBandwidth = kp.second;
            bestKernelParams.BLOCK_THREADS = bs;
            bestKernelParams.POINTS_PER_THREAD = ppt;
        }

        if (g_drawAllGraphs)
        {
            if (graphData.empty() || graphData.back().first != bs)
            {
                graphData.emplace_back(graphLineDataT(bs, std::vector<T>{float(ppt), kp.second}));
            } 
            else 
            {
                graphData.back().second.push_back(ppt);
                graphData.back().second.push_back(kp.second);
            }
        }
    };

#define runTest(bs, ppt) processResult(bs, ppt, runTileProcessing<bs, ppt, DIM, LOAD_MODIFIER, STORE_MODIFIER, MEM_LAYOUT, PRIVATE_MEM_LAYOUT, IO_BACKEND>(rdp, d_in, d_out, h_in, offset, stride));
    
    // runTest(64, 1);
    // runTest(64, 2);
    // runTest(64, 3);
    // runTest(64, 4);
    // runTest(64, 5);
    // runTest(64, 6);
    // runTest(64, 7);
    // runTest(64, 8);
    // runTest(64, 9);
    // runTest(64, 10);

    // runTest(96, 1);
    // runTest(96, 2);
    // runTest(96, 3);
    // runTest(96, 4);
    // runTest(96, 5);
    // runTest(96, 6);
    // runTest(96, 7);
    // runTest(96, 8);
    // runTest(96, 9);
    // runTest(96, 10);

    // runTest(128, 1);
    // runTest(128, 2);
    // runTest(128, 3);
    runTest(128, 4);
    // runTest(128, 5);
    // runTest(128, 6);
    // runTest(128, 7);
    // runTest(128, 8);
    // runTest(128, 9);
    // runTest(128, 10);

    // runTest(256, 1);
    // runTest(256, 2);
    // runTest(256, 3);
    // runTest(256, 4);
    // runTest(256, 5);
    // runTest(256, 6);
    // runTest(256, 7);
    // runTest(256, 8);
    // runTest(256, 9);
    // runTest(256, 10);

#undef runTest

    if (rdp.verbose)
    {
        *g_logFile << "% best performance conf: " << bestKernelParams.BLOCK_THREADS 
                    << ", " << bestKernelParams.POINTS_PER_THREAD 
                    << ", " << bestKernelParams.avgMillis
                    << ", " << bestKernelParams.gigaBandwidth << "\n"; 
    }

    if (g_drawAllGraphs)
    {
        std::ostringstream graphName;
        graphName << typeid(T).name() << DIM 
            << "__" << rd::LoadModifierNameTraits<LOAD_MODIFIER>::name
            << "__" << rd::StoreModifierNameTraits<STORE_MODIFIER>::name
            << "__" << rd::DataMemoryLayoutNameTraits<MEM_LAYOUT>::name
            << "__" << rd::DataMemoryLayoutNameTraits<PRIVATE_MEM_LAYOUT>::name
            << "__" << rd::BlockTileIONameTraits<IO_BACKEND>::name
            << "__" << rdp.np << "p";

        rd::GraphDrawer<T> gDrawer;

        gDrawer.setXLabel("Liczba punktów na wątek.");
        gDrawer.setYLabel("GB/s");
        gDrawer.showLegend();
        // format %.0f means 0 digits after the decimal point to print
        gDrawer.sendCmd("set format x '%.0f'");

        gDrawer.startGraph(graphName.str());
        for (size_t k = 0; k < graphData.size(); ++k)
        {
            auto &graphLine = graphData[k];
            std::ostringstream cmd;
            // ($1*10) -> multiplies x values by 10
            // :xticlabels(1) -> use values from first column as x tic's labels
            // :xtic(1) has the same meaning
            cmd << " '-' u ($1*10):2:xtic(1) t 'rozmiar bloku: " << graphLine.first << "' w lp ls " << (k+1) % gDrawer.stylesCnt << " ";
            gDrawer.addPlotCmd(cmd.str(), graphLine.second.data(), rd::GraphDrawer<T>::LINE, graphLine.second.size() / 2);
        }

        gDrawer.endGraph();
    }

    std::cout << ">>>>>>> best performance conf: " << bestKernelParams.BLOCK_THREADS 
                << ", " << bestKernelParams.POINTS_PER_THREAD 
                << ", " << bestKernelParams.avgMillis
                << ", " << bestKernelParams.gigaBandwidth; 

    return bestKernelParams;
}

//------------------------------------------------------------
//  TEST SPECIFIED VARIANTS
//------------------------------------------------------------

template <int DIM, typename T>
void test(rd::RDParams<T> &rdp,
          rd::RDSpiralParams<T> &rds)
{
    std::vector<std::string> samplesDir{"../../examples/data/nd_segments/", "../../examples/data/spirals/"};
    rd::gpu::Samples<T> d_samplesSet(rdp, rds, samplesDir, DIM);

    std::cout << "Samples: " << std::endl;
    std::cout <<  "\t dimension: " << rdp.dim << std::endl;
    std::cout <<  "\t n_samples: " << rdp.np << std::endl;

    std::cout << "Spiral params: " << std::endl;
    if (DIM == 2 || DIM == 3) 
    {
        std::cout <<  "\t a: " << rds.a << std::endl;
        std::cout <<  "\t b: " << rds.b << std::endl;
    }
    else
    {
        std::cout <<  "\t seg length: " << rds.a << std::endl;
    }
    std::cout <<  "\t sigma: " << rds.sigma << std::endl; 

    T *d_InRowMajor, *d_InColMajor;
    T *d_OutRowMajor, *d_OutColMajor;

    // allocate containers
    checkCudaErrors(hipMalloc((void**)&d_InRowMajor, rdp.np * DIM * sizeof(T)));
    checkCudaErrors(hipMalloc((void**)&d_InColMajor, rdp.np * DIM * sizeof(T)));
    checkCudaErrors(hipMalloc((void**)&d_OutRowMajor, rdp.np * DIM * sizeof(T)));
    checkCudaErrors(hipMalloc((void**)&d_OutColMajor, rdp.np * DIM * sizeof(T)));

    T *h_inRowMajor = new T[rdp.np * DIM];
    T *h_inColMajor = new T[rdp.np * DIM];

    // initialize data
    checkCudaErrors(hipMemcpy(d_InRowMajor, d_samplesSet.samples_, rdp.np * DIM * sizeof(T), hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(h_inRowMajor, d_samplesSet.samples_, rdp.np * DIM * sizeof(T), hipMemcpyDeviceToHost));

    rd::transposeTable(h_inRowMajor, h_inColMajor, rdp.np, DIM);
    checkCudaErrors(hipMemcpy(d_InColMajor, h_inColMajor, rdp.np * DIM * sizeof(T), hipMemcpyHostToDevice));
    checkCudaErrors(hipDeviceSynchronize());

    //---------------------------------------------------
    // Prepare logFile if needed
    //---------------------------------------------------

    if (rdp.verbose)
    {
        std::ostringstream logFileName;
        // append device name to log file
        logFileName << g_devName << "_" << std::to_string(DIM) <<
             "D" << LOG_FILE_NAME_SUFFIX;

        std::string logFilePath = rd::findPath("timings/", logFileName.str());
        g_logFile = new std::ofstream(logFilePath.c_str(), std::ios::out | std::ios::app);
        if (g_logFile->fail())
        {
            throw std::logic_error("Couldn't open file: " + logFileName.str());
        }

        *g_logFile << "%" << rd::HLINE << std::endl;
        *g_logFile << "% " << typeid(T).name() << std::endl;
        *g_logFile << "%" << rd::HLINE << std::endl;
    }

    //---------------------------------------------------
    //               GPU LOAD & WRITE
    //---------------------------------------------------

    std::vector<KernelParametersConf> bestConfigurations;
    KernelParametersConf bestKernelParams;

    auto processResults = [&bestKernelParams](int graphSec, KernelParametersConf params)
    {
        if (params.gigaBandwidth > bestKernelParams.gigaBandwidth)
        {
            bestKernelParams = params;
        }
        if (g_drawGraphs)
        {
            g_bestPerf[graphSec].push_back(DIM);
            g_bestPerf[graphSec].push_back(params.gigaBandwidth);
        }
    };

    std::cout << rd::HLINE << "\n";

    bestKernelParams =  testBlockPointsPerThreadConf<DIM, hipcub::LOAD_LDG, hipcub::STORE_DEFAULT, rd::ROW_MAJOR, rd::ROW_MAJOR, rd::gpu::IO_BACKEND_CUB>(rdp, d_InRowMajor, d_OutRowMajor, h_inRowMajor, int(0), int(1));
    if (g_drawGraphs)
    {
        g_bestPerf[0].push_back(DIM);
        g_bestPerf[0].push_back(bestKernelParams.gigaBandwidth);
    }

    processResults(1, testBlockPointsPerThreadConf<DIM, hipcub::LOAD_LDG, hipcub::STORE_DEFAULT, rd::ROW_MAJOR, rd::COL_MAJOR, rd::gpu::IO_BACKEND_CUB>(rdp, d_InRowMajor, d_OutRowMajor, h_inRowMajor, int(0), int(1)));
    processResults(2, testBlockPointsPerThreadConf<DIM, hipcub::LOAD_LDG, hipcub::STORE_DEFAULT, rd::COL_MAJOR, rd::COL_MAJOR, rd::gpu::IO_BACKEND_CUB>(rdp, d_InColMajor, d_OutColMajor, h_inColMajor, int(0), int(rdp.np)));
    processResults(3, testBlockPointsPerThreadConf<DIM, hipcub::LOAD_LDG, hipcub::STORE_DEFAULT, rd::COL_MAJOR, rd::ROW_MAJOR, rd::gpu::IO_BACKEND_CUB>(rdp, d_InColMajor, d_OutColMajor, h_inColMajor, int(0), int(rdp.np)));

    processResults(4, testBlockPointsPerThreadConf<DIM, hipcub::LOAD_LDG, hipcub::STORE_DEFAULT, rd::ROW_MAJOR, rd::ROW_MAJOR, rd::gpu::IO_BACKEND_TROVE>(rdp, d_InRowMajor, d_OutRowMajor, h_inRowMajor, int(0), int(1)));
    processResults(5, testBlockPointsPerThreadConf<DIM, hipcub::LOAD_LDG, hipcub::STORE_DEFAULT, rd::ROW_MAJOR, rd::COL_MAJOR, rd::gpu::IO_BACKEND_TROVE>(rdp, d_InRowMajor, d_OutRowMajor, h_inRowMajor, int(0), int(1)));
    processResults(6, testBlockPointsPerThreadConf<DIM, hipcub::LOAD_LDG, hipcub::STORE_DEFAULT, rd::COL_MAJOR, rd::COL_MAJOR, rd::gpu::IO_BACKEND_TROVE>(rdp, d_InColMajor, d_OutColMajor, h_inColMajor, int(0), int(rdp.np)));
    processResults(7, testBlockPointsPerThreadConf<DIM, hipcub::LOAD_LDG, hipcub::STORE_DEFAULT, rd::COL_MAJOR, rd::ROW_MAJOR, rd::gpu::IO_BACKEND_TROVE>(rdp, d_InColMajor, d_OutColMajor, h_inColMajor, int(0), int(rdp.np)));

    std::cout << rd::HLINE << "\n";

    //---------------------------------------------------
    //  summarize results
    //---------------------------------------------------

    if (rdp.verbose)
    {
        *g_logFile << "\n% overall best conf: " 
            << "\n%avgMillis: \t\t" << bestKernelParams.avgMillis
            << "\n%gigaBandwidth: \t" << bestKernelParams.gigaBandwidth
            << "\n%block threads: \t" << bestKernelParams.BLOCK_THREADS
            << "\n%points per thread: \t" << bestKernelParams.POINTS_PER_THREAD
            << "\n%load modifier: \t" << rd::getLoadModifierName(bestKernelParams.LOAD_MODIFIER)
            << "\n%store modifier: \t" << rd::getStoreModifierName(bestKernelParams.STORE_MODIFIER)
            << "\n%mem layout: \t\t" << rd::getRDDataMemoryLayout(bestKernelParams.MEM_LAYOUT)
            << "\n%priv mem layout: \t" << rd::getRDDataMemoryLayout(bestKernelParams.PRIVATE_MEM_LAYOUT)
            << "\n%io backend: \t\t" << rd::getRDTileIOBackend(bestKernelParams.IO_BACKEND)
            << "\n%numPoints: \t\t" << rdp.np << "\n";
    }

    std::cout << ">>>>> overall best conf: \n%" 
        << "\n avgMillis: \t\t" << bestKernelParams.avgMillis
        << "\n gigaBandwidth: \t" << bestKernelParams.gigaBandwidth
        << "\n block threads: \t" << bestKernelParams.BLOCK_THREADS
        << "\n points per thread: \t" << bestKernelParams.POINTS_PER_THREAD
        << "\n load modifier: \t" << rd::getLoadModifierName(bestKernelParams.LOAD_MODIFIER)
        << "\n store modifier: \t" << rd::getStoreModifierName(bestKernelParams.STORE_MODIFIER)
        << "\n mem layout: \t\t" << rd::getRDDataMemoryLayout(bestKernelParams.MEM_LAYOUT)
        << "\n priv mem layout: \t" << rd::getRDDataMemoryLayout(bestKernelParams.PRIVATE_MEM_LAYOUT)
        << "\n io backend: \t\t" << rd::getRDTileIOBackend(bestKernelParams.IO_BACKEND)
        << "\n numPoints: \t\t" << rdp.np << "\n";

    //---------------------------------------------------
    // clean-up
    
    if (rdp.verbose)
    {
        g_logFile->close();
        delete g_logFile;
    }

    delete[] h_inRowMajor;
    delete[] h_inColMajor;

    checkCudaErrors(hipFree(d_InRowMajor));
    checkCudaErrors(hipFree(d_InColMajor));
    checkCudaErrors(hipFree(d_OutRowMajor));
    checkCudaErrors(hipFree(d_OutColMajor));
}

template <typename T>
std::string createFinalGraphDataFile()
{
    //------------------------------------------
    // create data file for drawing graph
    //------------------------------------------

    std::ostringstream graphDataFile;
    graphDataFile << typeid(T).name() << "_" << g_devName << "_graphData_v4.dat";

    std::string filePath = rd::findPath("gnuplot_data/", graphDataFile.str());
    std::ofstream gdataFile(filePath.c_str(), std::ios::out | std::ios::trunc);
    if (gdataFile.fail())
    {
        throw std::logic_error("Couldn't open file: " + graphDataFile.str());
    }

    auto printData = [&gdataFile](std::vector<float> const &v, std::string secName)
    {
        gdataFile << "# [" << secName << "] \n";
        for (size_t i = 0; i < v.size()/2; ++i)
        {
            gdataFile << std::right << std::fixed << std::setw(5) << std::setprecision(1) <<
                v[2 * i] << " " << v[2 * i + 1] << "\n";
        }
        // two sequential blank records to reset $0 counter
        gdataFile << "\n\n";
    };

    printData(g_bestPerf[0], "ROW-ROW-(CUB)");
    printData(g_bestPerf[1], "ROW-COL-(CUB)");
    printData(g_bestPerf[2], "COL-COL-(CUB)");
    printData(g_bestPerf[3], "COL-ROW-(CUB)");
    printData(g_bestPerf[4], "ROW-ROW-(trove)");
    printData(g_bestPerf[5], "ROW-COL-(trove)");
    printData(g_bestPerf[6], "COL-COL-(trove)");
    printData(g_bestPerf[7], "COL-ROW-(trove)");

    gdataFile.close();
    return filePath;
}

template <typename T>
void drawFinalGraph(std::string graphDataFilePath)
{
        //------------------------------------------
        // drawing graph
        //------------------------------------------

        rd::GraphDrawer<float> gDrawer;
        std::ostringstream graphName;
        graphName << typeid(T).name() << "_" << g_devName << "_bandwidths_v4.png";
        std::string filePath = rd::findPath("img/", graphName.str());

        gDrawer.sendCmd("set output '" + filePath + "'");
        gDrawer.setXLabel("Wymiar danych.");
        gDrawer.setYLabel("GB/s");

        gDrawer.sendCmd("set key right top");
        gDrawer.sendCmd("set style fill solid 0.95 border rgb 'grey30'");

        gDrawer.sendCmd("colStep = " + std::to_string(g_graphColStep));
        gDrawer.sendCmd("bs = 2 * colStep");
        gDrawer.sendCmd("nCol = " + std::to_string(g_graphNCol));
        gDrawer.sendCmd("groupStep = (nCol+1) * bs");
        gDrawer.sendCmd("nGroups = " + std::to_string(g_graphNGroups));
        gDrawer.sendCmd("offset = 9 * colStep");
        gDrawer.sendCmd("xEnd = offset + (nGroups-1) * groupStep + 9 * colStep + 4");

        gDrawer.sendCmd("set xrange [0:xEnd]");
        gDrawer.sendCmd("set xtics nomirror out ('2D' offset,'3D' offset + groupStep,"
             "'4D' offset + 2*groupStep, '5D' offset + 3*groupStep, '6D' offset + 4*groupStep)");
        gDrawer.sendCmd("dataFile = '" + graphDataFilePath + "'");

        std::ostringstream cmd;
        cmd << "plot dataFile i 0 u (offset + $0 * groupStep - 7 * colStep):2:(bs) t 'ROW-ROW (CUB)' w boxes ls 1,";
        cmd << "    ''        i 1 u (offset + $0 * groupStep - 5 * colStep):2:(bs) t 'ROW-COL (CUB)' w boxes ls 2,";
        cmd << "    ''        i 2 u (offset + $0 * groupStep - 3 * colStep):2:(bs) t 'COL-COL (CUB)' w boxes ls 3,";
        cmd << "    ''        i 3 u (offset + $0 * groupStep - 1 * colStep):2:(bs) t 'COL-ROW (CUB)' w boxes ls 4,";
        cmd << "    ''        i 4 u (offset + $0 * groupStep + 1 * colStep):2:(bs) t 'ROW-ROW (trove)' w boxes ls 5,";
        cmd << "    ''        i 5 u (offset + $0 * groupStep + 3 * colStep):2:(bs) t 'ROW-COL (trove)' w boxes ls 6,";
        cmd << "    ''        i 6 u (offset + $0 * groupStep + 5 * colStep):2:(bs) t 'COL-COL (trove)' w boxes ls 7,";
        cmd << "    ''        i 7 u (offset + $0 * groupStep + 7 * colStep):2:(bs) t 'COL-ROW (trove)' w boxes ls 8,";
        cmd << "    ''        i 0 u (offset + $0 * groupStep - 7 * colStep):($2 + 0.6):2 notitle w labels rotate by 70 left,";
        cmd << "    ''        i 1 u (offset + $0 * groupStep - 5 * colStep):($2 + 0.6):2 notitle w labels rotate by 70 left,";
        cmd << "    ''        i 2 u (offset + $0 * groupStep - 3 * colStep):($2 + 0.6):2 notitle w labels rotate by 70 left,";
        cmd << "    ''        i 3 u (offset + $0 * groupStep - 1 * colStep):($2 + 0.6):2 notitle w labels rotate by 70 left,";
        cmd << "    ''        i 4 u (offset + $0 * groupStep + 1 * colStep):($2 + 0.6):2 notitle w labels rotate by 70 left,";
        cmd << "    ''        i 5 u (offset + $0 * groupStep + 3 * colStep):($2 + 0.6):2 notitle w labels rotate by 70 left,";
        cmd << "    ''        i 6 u (offset + $0 * groupStep + 5 * colStep):($2 + 0.6):2 notitle w labels rotate by 70 left,";
        cmd << "    ''        i 7 u (offset + $0 * groupStep + 7 * colStep):($2 + 0.6):2 notitle w labels rotate by 70 left ";

        gDrawer.sendCmd(cmd.str());
}

int main(int argc, char const **argv)
{

    rd::RDParams<double> dParams;
    rd::RDSpiralParams<double> dSParams;
    rd::RDParams<float> fParams;
    rd::RDSpiralParams<float> fSParams;

    //-----------------------------------------------------------------

    // Initialize command line
    rd::CommandLineArgs args(argc, argv);
    if (args.CheckCmdLineFlag("help")) 
    {
        printf("%s \n"
            "\t\t[--np=<P size>]\n"
            "\t\t[--a=<spiral param>]\n"
            "\t\t[--b=<spiral param>]\n"
            "\t\t[--s=<spiral noise sigma>]\n"
            "\t\t[--d=<device id>]\n"
            "\t\t[--v <verbose>]\n"
            "\t\t[--f=<file name to load>]\n"
            "\t\t[--g <draw graphs>]\n"
            "\t\t[--ga <draw all graphs (a lot)>]\n"
            "\n", argv[0]);
        exit(0);
    }

    if (args.CheckCmdLineFlag("f"))
    {
        args.GetCmdLineArgument("f", fSParams.file);
        args.GetCmdLineArgument("f", dSParams.file);
        fSParams.loadFromFile = true;
        dSParams.loadFromFile = true;
    }
    else
    {
        args.GetCmdLineArgument("np", dParams.np);
        args.GetCmdLineArgument("np", fParams.np);

        if (args.CheckCmdLineFlag("a")) 
        {
            args.GetCmdLineArgument("a", fSParams.a);
            args.GetCmdLineArgument("a", dSParams.a);
        }
        if (args.CheckCmdLineFlag("b")) 
        {
            args.GetCmdLineArgument("b", fSParams.b);
            args.GetCmdLineArgument("b", dSParams.b);
        }
        if (args.CheckCmdLineFlag("s")) 
        {
            args.GetCmdLineArgument("s", fSParams.sigma);
            args.GetCmdLineArgument("s", dSParams.sigma);
        }
    }
    if (args.CheckCmdLineFlag("d")) 
    {
        args.GetCmdLineArgument("d", fParams.devId);
        args.GetCmdLineArgument("d", dParams.devId);
    }
    if (args.CheckCmdLineFlag("v")) 
    {
        fParams.verbose = true;
        dParams.verbose = true;
    }
    if (args.CheckCmdLineFlag("g")) 
    {
        g_drawGraphs = true;
    }
    if (args.CheckCmdLineFlag("ga")) 
    {
        g_drawAllGraphs = true;
        g_drawGraphs = true;
    }

    checkCudaErrors(deviceInit(fParams.devId));

    // set device name for logging and drawing purposes
    fParams.devId = (fParams.devId != -1) ? fParams.devId : 0;
    hipDeviceProp_t devProp;
    checkCudaErrors(hipGetDeviceProperties(&devProp, fParams.devId));
    g_devName = devProp.name;

    if (g_drawGraphs)
    {
        // initialize storage for graph data
        g_bestPerf = std::vector<std::vector<float>>(8);
    }

    //-----------------------------------------
    //  TESTS
    //-----------------------------------------

    std::cout << rd::HLINE << std::endl;
    std::cout << "FLOAT 2D: " << std::endl;
    test<2>(fParams, fSParams);
    std::cout << rd::HLINE << std::endl;
    std::cout << "FLOAT 3D: " << std::endl;
    test<3>(fParams, fSParams);
    std::cout << rd::HLINE << std::endl;
    std::cout << "FLOAT 4D: " << std::endl;
    test<4>(fParams, fSParams);
    std::cout << rd::HLINE << std::endl;
    std::cout << "FLOAT 5D: " << std::endl;
    test<5>(fParams, fSParams);
    std::cout << rd::HLINE << std::endl;
    std::cout << "FLOAT 6D: " << std::endl;
    test<6>(fParams, fSParams);
    std::cout << rd::HLINE << std::endl;

    if (g_drawGraphs)
    {
        drawFinalGraph<float>(createFinalGraphDataFile<float>());

        g_bestPerf.clear();
        g_bestPerf = std::vector<std::vector<float>>(8);
    }

    // std::cout << "DOUBLE 2D: " << std::endl;
    // test<2>(dParams, dSParams);
    // std::cout << rd::HLINE << std::endl;
    // std::cout << "DOUBLE 3D: " << std::endl;
    // test<3>(dParams, dSParams);
    // std::cout << rd::HLINE << std::endl;
    // std::cout << "DOUBLE 4D: " << std::endl;
    // test<4>(dParams, dSParams);
    // std::cout << rd::HLINE << std::endl;
    // std::cout << "DOUBLE 5D: " << std::endl;
    // test<5>(dParams, dSParams);
    // std::cout << rd::HLINE << std::endl;
    // std::cout << "DOUBLE 6D: " << std::endl;
    // test<6>(dParams, dSParams);
    // std::cout << rd::HLINE << std::endl;

    // if (g_drawGraphs)
    // {
    //     drawFinalGraph<double>(createFinalGraphDataFile<double>());

    //     g_bestPerf.clear();
    // }

    checkCudaErrors(deviceReset());

    std::cout << "END!" << std::endl;
    return 0;
}
