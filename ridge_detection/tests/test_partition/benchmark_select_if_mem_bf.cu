#include "hip/hip_runtime.h"
/**
 * @file benchmark_select_if_mem_bf.cu
 * @author     Adam Rogowiec
 *
 * This file is an integral part of the master thesis entitled:
 * "Elaboration and implementation in CUDA technology parallel version of
 * estimation of multidimensional random variable density function ridge
 * detection algorithm."
 * , which is conducted under the supervision of prof. dr hab. inż. Marek
 * Nałęcz.
 *
 * Institute of Control and Computation Engineering Faculty of Electronics and
 * Information Technology Warsaw University of Technology 2016
 */

#define BLOCK_TILE_LOAD_V4 1

#include "rd/utils/graph_drawer.hpp"
#include "rd/utils/cmd_line_parser.hpp"
#include "rd/utils/utilities.hpp"
#include "rd/utils/memory.h" 
#include "rd/utils/name_traits.hpp"
#include "rd/utils/rd_params.hpp"

#include "rd/gpu/block/block_select_if.cuh"
#include "rd/gpu/util/dev_samples_set.cuh"
#include "rd/gpu/util/dev_utilities.cuh"
#include "rd/gpu/util/dev_memcpy.cuh"

#include "tests/test_util.hpp"
#include "cub/test_util.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <sstream>
#include <fstream>
#include <iomanip>
#include <typeinfo>
#include <stdexcept>
#include <string>
#include <limits>

#include <cmath>
#include <functional>
#include <algorithm>
#include <utility>
#include <tuple>

#ifdef RD_USE_OPENMP
#include <omp.h>
#endif

//------------------------------------------------------------
//  GLOBAL CONSTANTS / VARIABLES
//------------------------------------------------------------

static const std::string LOG_FILE_NAME_SUFFIX = "_select_if_mbf-timings.txt";

std::ofstream * g_logFile       = nullptr;
bool            g_logResults    = false;
bool            g_drawGraphs    = false;
std::string     g_devName;
int             g_devSMCount        = 0;
int             g_devMaxBlocksPerSM = 0;
bool            g_verbose           = false;

// forward declaration
struct KernelParametersConf;

// (selectRatio, DIM, kernelConf)
typedef std::tuple<float, unsigned int, KernelParametersConf> GraphDataT;

std::vector<GraphDataT>         g_graphData;

#if defined(RD_PROFILE) || defined(RD_DEBUG)
static const int g_iterations = 1;
#else
static const int g_iterations = 100;
#endif

//------------------------------------------------------------
//  Benchmark helper structures
//------------------------------------------------------------

struct KernelParametersConf
{
    int                         BLOCK_THREADS;
    int                         POINTS_PER_THREAD;
    int                         DIM;
    hipcub::CacheLoadModifier      LOAD_MODIFIER;
    rd::gpu::BlockTileIOBackend IO_BACKEND;
    rd::DataMemoryLayout        INPUT_MEM_LAYOUT;
    bool                        STORE_TWO_PHASE;
    float                       avgMillis;
    float                       gigaBandwidth;

    KernelParametersConf()
    :
        BLOCK_THREADS(0),
        POINTS_PER_THREAD(0),
        DIM(0),
        LOAD_MODIFIER(hipcub::LOAD_DEFAULT),
        INPUT_MEM_LAYOUT(rd::ROW_MAJOR),
        IO_BACKEND(rd::gpu::IO_BACKEND_CUB),
        STORE_TWO_PHASE(false),
        avgMillis(std::numeric_limits<float>::max()),
        gigaBandwidth(std::numeric_limits<float>::lowest())
    {}

    KernelParametersConf(
        int                         _DIM,
        hipcub::CacheLoadModifier      _LOAD_MODIFIER,
        rd::gpu::BlockTileIOBackend _IO_BACKEND,
        rd::DataMemoryLayout        _INPUT_MEM_LAYOUT,
        bool                        _STORE_TWO_PHASE)
    :
        DIM(_DIM),
        LOAD_MODIFIER(_LOAD_MODIFIER),
        IO_BACKEND(_IO_BACKEND),
        INPUT_MEM_LAYOUT(_INPUT_MEM_LAYOUT),
        STORE_TWO_PHASE(_STORE_TWO_PHASE)
    {}

    void printLaunchConf(std::ostream& os) const
    {
        os << POINTS_PER_THREAD 
            << " " << BLOCK_THREADS 
            << " " << avgMillis
            << " " << gigaBandwidth << "\n";
    }

    bool hasSameAlgParams(KernelParametersConf const & kpc) const
    {
        return  LOAD_MODIFIER       == kpc.LOAD_MODIFIER &&
                IO_BACKEND          == kpc.IO_BACKEND &&
                INPUT_MEM_LAYOUT    == kpc.INPUT_MEM_LAYOUT &&
                STORE_TWO_PHASE     == kpc.STORE_TWO_PHASE;
    }
    
    KernelParametersConf getAlgParamCopy() const
    {
        KernelParametersConf out = *this;
        out.BLOCK_THREADS = 0;
        out.POINTS_PER_THREAD = 0;
        out.avgMillis = 0.f;
        out.gigaBandwidth = 0.f;
        return out;
    }
};

std::ostream & operator<<(std::ostream & os, KernelParametersConf const & kp)
{
    os << "\n dim: \t\t\t" << kp.DIM
        << "\n avgMillis: \t\t" << kp.avgMillis
        << "\n gigaBandwidth: \t" << kp.gigaBandwidth
        << "\n block threads: \t" << kp.BLOCK_THREADS
        << "\n points per thread: \t" << kp.POINTS_PER_THREAD
        << "\n load modifier: \t" << rd::getLoadModifierName(kp.LOAD_MODIFIER)
        << "\n mem layout: \t\t" << rd::getRDDataMemoryLayout(kp.INPUT_MEM_LAYOUT)
        << "\n io backend: \t\t" << rd::getRDTileIOBackend(kp.IO_BACKEND)
        << "\n storeTwoPhase \t\t" << std::boolalpha << kp.STORE_TWO_PHASE << "\n";
    return os;
}

typedef std::pair<float, float> KernelPerfT;

//------------------------------------------------------------
//  Gnuplot data file generation
//------------------------------------------------------------


template <typename T>
std::string createFinalGraphDataFile(
    unsigned int MIN_TEST_DIM,
    unsigned int MAX_TEST_DIM,
    int numPoints)
{
    // assume that all current elements in graphData have the same selecRatio.
    float selectRatio = std::get<0>(g_graphData[0]);

    std::ostringstream graphDataFileName;
    graphDataFileName << typeid(T).name() << "_" << g_devName 
        << "_selRatio" << selectRatio 
        << "_nPoints" << numPoints
        << "_graphData.dat";

    std::string filePath = rd::findPath("gnuplot_data/", graphDataFileName.str());
    std::ofstream gdataFile(filePath.c_str(), std::ios::out | std::ios::trunc);
    if (gdataFile.fail())
    {
        throw std::logic_error("Couldn't open file: " + graphDataFileName.str());
    }

    /**
     * group graph data into graph columns (readable form for gnuplot)
     * g_graphData has structure:
     * [0] DIM 1 (alg param set1)
     * [2] DIM 1 (alg param set2)
     * [3] DIM 1 (alg param set3)
     * [4] DIM 1 (alg param set4)
     *  ...
     * [n] DIM 2 (alg param set1)
     * [n+1] DIM 2 (alg param set2)
     * [n+2] DIM 2 (alg param set3)
     * [n+3] DIM 2 (alg param set4)
     *  ...
     */

    std::vector<std::pair<KernelParametersConf, std::vector<float>>> graphColumns;

    // read first dimension
    for (auto it = g_graphData.begin(); it != g_graphData.end();)
    {
        if (std::get<1>(*it) == MIN_TEST_DIM)
        {
            // add column
            KernelParametersConf tmp = std::get<2>(*it);
            graphColumns.emplace_back(tmp.getAlgParamCopy(), std::vector<float>{float(MIN_TEST_DIM), tmp.gigaBandwidth});
            // erase element from g_graphData
            it = g_graphData.erase(it);
        }
        else
        {
            break;
        }
    }

    // read other dimension
    for (unsigned int dim = MIN_TEST_DIM+1; dim <= MAX_TEST_DIM; ++dim)
    {
        unsigned int column = 0;
        for (auto it = g_graphData.begin(); it != g_graphData.end();)
        {
            if (std::get<1>(*it) == dim)
            {
                KernelParametersConf tmp = std::get<2>(*it);
                if (!tmp.hasSameAlgParams(graphColumns[column].first))
                {
                    std::cerr << "ERROR! incorrect columns order!" << std::endl;
                    std::cout << "\n tmp: \n" << tmp << std::endl;
                    std::cout << "\n graphColumns["<<column<<"]: \n" << graphColumns[column].first << std::endl;
                    exit(1);
                }
                graphColumns[column].second.push_back(dim);
                graphColumns[column].second.push_back(tmp.gigaBandwidth);
                // erase element from g_graphData
                it = g_graphData.erase(it);
                column++;
            }
            else
            {
                break;
            }
        }
    }

    for (auto const & e : graphColumns)
    {
        // prepare secname
        std::ostringstream secName;
        secName << rd::getLoadModifierName(e.first.LOAD_MODIFIER)
                << "_" << rd::getRDDataMemoryLayout(e.first.INPUT_MEM_LAYOUT)
                << "_" << rd::getRDTileIOBackend(e.first.IO_BACKEND)
                << "_" << std::boolalpha << e.first.STORE_TWO_PHASE;

        auto v = e.second;

        gdataFile << "# [" << secName.str() << "] \n";
        for (size_t i = 0; i < v.size() / 2; ++i)
        {
            gdataFile << std::right << std::fixed << std::setw(5) << std::setprecision(1) <<
                int(v[2*i]) << " " << v[2*i + 1] << "\n";
        }
        // two sequential blank records to reset $0 counter
        gdataFile << "\n\n";

    }

    gdataFile.close();
    return filePath;
}

//------------------------------------------------------------
//  Select Op
//------------------------------------------------------------

template <int DIM, typename T>
struct LessThan
{
    T val_;

    __host__ __device__ __forceinline__ LessThan(T v)
    : 
        val_(v)
    {}

    __host__ __device__ __forceinline__ bool operator()(T const * point) const
    {
        for (int d = 0; d < DIM; ++d)
        {
            if (point[d] >= val_) 
                return false;
        }
        return true;
    }
};

//------------------------------------------------------------
//  KERNEL 
//------------------------------------------------------------

template <
    typename                    BlockSelectIfPolicyT,
    int                         DIM,
    rd::DataMemoryLayout        INPUT_MEM_LAYOUT,
    typename                    OffsetT,
    typename                    SampleT,
    typename                    SelectOpT,
    bool                        STORE_TWO_PHASE>    // Whether or not to perform two phase selected items store with items compatcion in shmem. Otherwise uses warp-wide store.
__launch_bounds__ (int(BlockSelectIfPolicyT::BLOCK_THREADS))
static __global__ void selectIfKernel(
    SampleT const *                         d_in,
    OffsetT                                 numPoints,
    SampleT **                              d_selectedItemsPtrs,
    OffsetT *                               d_selectedItemsCnt,
    SelectOpT                               selectOp,
    OffsetT                                 inStride,
    OffsetT                                 outStride)
{
    typedef rd::gpu::BlockSelectIf<
        BlockSelectIfPolicyT,
        DIM,
        INPUT_MEM_LAYOUT,
        SelectOpT,
        SampleT,
        OffsetT,
        STORE_TWO_PHASE>
    BlockSelectIfT;

    if (numPoints == 0)
    {
        return;
    }

    __shared__ typename BlockSelectIfT::TempStorage tempStorage;

    OffsetT selectedPointsCnt = BlockSelectIfT(tempStorage, d_in, d_selectedItemsPtrs[blockIdx.x], 
        selectOp).scanRange(0, numPoints, inStride, outStride);

    if (threadIdx.x == 0)
    {
        d_selectedItemsCnt[blockIdx.x] = selectedPointsCnt;
    }
}

//------------------------------------------------------------
//  KERNEL INVOCATION
//------------------------------------------------------------

struct KernelConfig
{
    int blockThreads;
    int itemsPerThread;
};

template <
    typename                    OffsetT,
    typename                    SampleT,
    typename                    SelectOpT,
    typename                    PartitionKernelPtrT>
static hipError_t invoke(
    SampleT const *                 d_in,
    OffsetT                         numPoints,
    SampleT **                      d_selectedItemsPtrs,
    OffsetT *                       d_selectedItemsCnt,
    OffsetT                         inStride,
    OffsetT                         outStride,
    SelectOpT                       selectOp,
    hipStream_t                    stream,
    bool                            debugSynchronous,
    PartitionKernelPtrT             partitionKernelPtr,
    KernelConfig                    kernelConfig)
{

    hipError_t error = hipSuccess;
    do
    {
        // get SM occupancy
        int smOccupancy;
        if(HipcubDebug(hipcub::MaxSmOccupancy(
            smOccupancy,
            partitionKernelPtr,
            kernelConfig.blockThreads)
        )) break;

        dim3 partitionGridSize(1);
        partitionGridSize.x = smOccupancy * g_devSMCount;

        if (debugSynchronous)
        {
            printf("Invoking selectIfKernel<<<%d, %d, 0, %lld>>> numPoints: %d, "
                "pointsPerThread: %d\n",
                partitionGridSize.x, kernelConfig.blockThreads, (long long)stream, numPoints, 
                kernelConfig.itemsPerThread);
        }

        partitionKernelPtr<<<partitionGridSize, kernelConfig.blockThreads, 0, stream>>>(
            d_in,
            numPoints,
            d_selectedItemsPtrs,
            d_selectedItemsCnt,
            selectOp,
            inStride,
            outStride);

        // Check for failure to launch
        if (HipcubDebug(error = hipPeekAtLastError())) break;
        // Sync the stream if specified to flush runtime errors
        if (debugSynchronous && (HipcubDebug(error = hipcub::SyncStream(stream)))) break;


    } while (0);

    return error;
}

//------------------------------------------------------------
//  KERNEL DISPATCH
//------------------------------------------------------------

template <
    int                         BLOCK_THREADS,
    int                         POINTS_PER_THREAD,
    hipcub::CacheLoadModifier      LOAD_MODIFIER,
    rd::gpu::BlockTileIOBackend IO_BACKEND,
    rd::DataMemoryLayout        INPUT_MEM_LAYOUT,
    int                         DIM,
    bool                        STORE_TWO_PHASE,
    typename                    SelectOpT,
    typename                    OffsetT,
    typename                    T>
static void dispatchKernel(
    T const *                       d_in,
    OffsetT                         numPoints,
    T **                            d_selectedItemsPtrs,
    OffsetT *                       d_selectedItemsCnt,
    SelectOpT                       selectOp,
    OffsetT                         inStride,
    OffsetT                         outStride,
    int                             iterations,
    bool                            debugSynchronous = false)
{

    typedef rd::gpu::BlockSelectIfPolicy<
        BLOCK_THREADS,
        POINTS_PER_THREAD,
        LOAD_MODIFIER,
        IO_BACKEND>
    BlockSelectIfPolicyT;

    KernelConfig partitionConfig;
    partitionConfig.blockThreads = BLOCK_THREADS;
    partitionConfig.itemsPerThread = POINTS_PER_THREAD;

    auto partitionKernelPtr = selectIfKernel<BlockSelectIfPolicyT, DIM, INPUT_MEM_LAYOUT,
        OffsetT, T, SelectOpT, STORE_TWO_PHASE>;

    // If we use two-phase store algorithm, which compact's selections in smem, we prefer larger smem to L1 cache size.
    if (STORE_TWO_PHASE)
    {
        // set smem/L1 mem configuration
        // * - ::hipFuncCachePreferNone: no preference for shared memory or L1 (default)
        // * - ::hipFuncCachePreferShared: prefer larger shared memory and smaller L1 cache
        // * - ::hipFuncCachePreferL1: prefer larger L1 cache and smaller shared memory
        // * - ::hipFuncCachePreferEqual: prefer equal size L1 cache and shared memory
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(partitionKernelPtr), hipFuncCachePreferShared);

        if (sizeof(T) == 8)
        {
            // * - ::hipSharedMemBankSizeDefault: use the device's shared memory configuration when launching this function.
            // * - ::hipSharedMemBankSizeFourByte: set shared memory bank width to be four bytes natively when launching this function.
            // * - ::hipSharedMemBankSizeEightByte: set shared memory bank width to be eight bytes natively when launching this function.
            hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(partitionKernelPtr), hipSharedMemBankSizeEightByte);
        }
    }

    for (int i = 0; i < iterations; ++i)
    {
        CubDebugExit(invoke(
            d_in,
            numPoints,
            d_selectedItemsPtrs,
            d_selectedItemsCnt,
            inStride,
            outStride,
            selectOp,
            0,
            debugSynchronous,
            partitionKernelPtr,
            partitionConfig));
    }

}

//------------------------------------------------------------
//  Test and benchmark specified kernel configuration 
//------------------------------------------------------------

template <
    int                         BLOCK_THREADS,
    int                         POINTS_PER_THREAD,
    int                         DIM,
    hipcub::CacheLoadModifier      LOAD_MODIFIER,
    rd::gpu::BlockTileIOBackend IO_BACKEND,
    rd::DataMemoryLayout        INPUT_MEM_LAYOUT,
    bool                        STORE_TWO_PHASE,
    typename                    T>
KernelPerfT runSelectIf(
    int                             numPoints,
    float                           compare,
    T const *                       d_in,
    int                             inStride,
    int                             outStride,
    T const *                       h_reference,
    int const                       h_referenceSelectedPointsCnt)
{
    std::cout << rd::HLINE << std::endl;
    std::cout << "runSelectIf:" << std::endl;
    std::cout << "blockThreads: " << BLOCK_THREADS 
              << ", pointsPerThread: " << POINTS_PER_THREAD
              << ", load modifier: " << rd::LoadModifierNameTraits<LOAD_MODIFIER>::name
              << ", io backend: " << rd::BlockTileIONameTraits<IO_BACKEND>::name
              << ", mem layout: " << rd::DataMemoryLayoutNameTraits<INPUT_MEM_LAYOUT>::name
              << ", store two phase: " << std::boolalpha << STORE_TWO_PHASE
              << ", numPoints: " << numPoints << "\n";

    LessThan<DIM, T> selectOp(compare);

    // type definitions for kernel function pointer
    typedef LessThan<DIM, T> SelectOpT;
    typedef rd::gpu::BlockSelectIfPolicy<
        BLOCK_THREADS,
        POINTS_PER_THREAD,
        LOAD_MODIFIER,
        IO_BACKEND>
    BlockSelectIfPolicyT;

    auto kernelPtr = selectIfKernel<BlockSelectIfPolicyT, DIM, INPUT_MEM_LAYOUT, int, T, 
                        SelectOpT, STORE_TWO_PHASE>;

    // get SM occupancy
    int smOccupancy;
    CubDebugExit(hipcub::MaxSmOccupancy(
        smOccupancy,
        kernelPtr,
        BLOCK_THREADS)
    );

    int blockCount = 1;
    blockCount = smOccupancy * g_devSMCount;

    // Allocate device arrays
    T ** d_selectedPointsPtrs = nullptr;
    T ** h_dSelectedPointsPtrs = new T*[blockCount];
    int * d_selectedPointsCnt = nullptr;

    checkCudaErrors(hipMalloc(&d_selectedPointsPtrs, blockCount * sizeof(T*)));
    checkCudaErrors(hipMalloc(&d_selectedPointsCnt, blockCount * sizeof(int)));
    
    hipStream_t auxStream;
    checkCudaErrors(hipStreamCreateWithFlags(&auxStream, hipStreamNonBlocking));
    checkCudaErrors(hipMemsetAsync(d_selectedPointsCnt, 0, blockCount * sizeof(int), auxStream));

    for (int k = 0; k < blockCount; ++k)
    {
        checkCudaErrors(hipMalloc(h_dSelectedPointsPtrs + k, 
            h_referenceSelectedPointsCnt * DIM * sizeof(T)));
        checkCudaErrors(hipMemsetAsync(h_dSelectedPointsPtrs[k], 0, 
            h_referenceSelectedPointsCnt * DIM * sizeof(T), auxStream));
    }

    // Initialize device input
    checkCudaErrors(hipMemcpy(d_selectedPointsPtrs, h_dSelectedPointsPtrs, 
        blockCount * sizeof(T*), hipMemcpyHostToDevice));
    checkCudaErrors(hipDeviceSynchronize());

    // Run warm-up/correctness iteration
    dispatchKernel<BLOCK_THREADS, POINTS_PER_THREAD, LOAD_MODIFIER, IO_BACKEND, INPUT_MEM_LAYOUT, 
        DIM, STORE_TWO_PHASE>(
            d_in, numPoints, d_selectedPointsPtrs, d_selectedPointsCnt, selectOp, inStride, 
            outStride, 1, true);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // check results
    std::cout << "\nCheck results count ... ";
    if (CompareDeviceResults(h_referenceSelectedPointsCnt, d_selectedPointsCnt, blockCount))
    {
        std::cerr << "\n\n ERROR! Incorrect results count!" << std::endl;
        exit(1);
    }
    else
    {
        std::cout << " PASS!\n";
    }

    std::cout << "Check each block results ";
    #ifdef RD_USE_OPENMP
    #pragma omp parallel for schedule(static)
    #endif
    for (int i = 0; i < blockCount; ++i)
    {
        // compareDeviceResults sorts device data
        if (CompareDeviceResults(h_reference, h_dSelectedPointsPtrs[i],
                h_referenceSelectedPointsCnt * DIM, true, true, false))
        {
            std::cerr << "\n\n ERROR! Incorrect device selected items!" << std::endl;
            exit(1);
        }
    }
    std::cout << " PASS!\n";


    // Measure performance
    GpuTimer timer;
    float elapsedMillis;

    #ifdef RD_PROFILE
    hipProfilerStart();
    #endif
    timer.Start();
    dispatchKernel<BLOCK_THREADS, POINTS_PER_THREAD, LOAD_MODIFIER, IO_BACKEND, INPUT_MEM_LAYOUT, 
        DIM, STORE_TWO_PHASE>(
            d_in, numPoints, d_selectedPointsPtrs, d_selectedPointsCnt, selectOp, inStride, 
            outStride, g_iterations);
    timer.Stop();
    elapsedMillis = timer.ElapsedMillis();
    #ifdef RD_PROFILE
    hipProfilerStop();
    #endif



    float   avgMillis           = elapsedMillis / g_iterations;
    int     selItemsCnt         = h_referenceSelectedPointsCnt * DIM * blockCount;
    size_t  numBytes            = sizeof(T) * blockCount * numPoints * DIM +              // every block scans entire data set
                                    blockCount * sizeof(int) +                            // storing selectedItems counters
                                    selItemsCnt * sizeof(T);                                      // storing selected items
    float   gigaBandwidth       =   float(numBytes) / avgMillis / 1000.0 / 1000.0;                    // conversion to GB/s

    if (g_logResults)
    {
        *g_logFile << POINTS_PER_THREAD << " " << BLOCK_THREADS << " " << avgMillis 
                    << " " << gigaBandwidth << "\n";
    }

    std::cout << avgMillis << " avg ms, "
              << gigaBandwidth << " logical GB/s\n";

    // cleanup
    checkCudaErrors(hipStreamDestroy(auxStream));
    for (int k = 0; k < blockCount; ++k)
    {
        if (h_dSelectedPointsPtrs[k]) checkCudaErrors(hipFree(h_dSelectedPointsPtrs[k]));
    }

    if (d_selectedPointsPtrs) checkCudaErrors(hipFree(d_selectedPointsPtrs));
    if (d_selectedPointsCnt) checkCudaErrors(hipFree(d_selectedPointsCnt));
    if (h_dSelectedPointsPtrs) delete[] h_dSelectedPointsPtrs;


    return std::make_pair(avgMillis, gigaBandwidth);
}

//------------------------------------------------------------
//  Test kernel block-threads / items-per-thread configurations
//------------------------------------------------------------

/*
 *  Test different kernel configurations (block size, points per thread)
 */
template <
    int                         DIM,
    hipcub::CacheLoadModifier      LOAD_MODIFIER,
    rd::gpu::BlockTileIOBackend IO_BACKEND,
    rd::DataMemoryLayout        INPUT_MEM_LAYOUT,
    bool                        STORE_TWO_PHASE,
    typename                    T>
KernelParametersConf testKernelConf(
    int                             numPoints,
    float                           compare,
    T const *                       d_in,
    int                             inStride,
    int                             outStride,
    T const *                       h_reference,
    int const                       h_referenceSelectedPointsCnt)
{

    KernelParametersConf bestKernelParams(DIM, LOAD_MODIFIER, IO_BACKEND, INPUT_MEM_LAYOUT, 
        STORE_TWO_PHASE);
    KernelPerfT bestPerf = std::make_pair(1e10f, -1.0f);

    auto checkBestConf = [&bestPerf, &bestKernelParams](int bs, int ppt, KernelPerfT kp)
    {
        if (kp.second > bestPerf.second)
        {
            bestPerf.first = kp.first;
            bestPerf.second = kp.second;
            bestKernelParams.avgMillis = kp.first;
            bestKernelParams.gigaBandwidth = kp.second;
            bestKernelParams.BLOCK_THREADS = bs;
            bestKernelParams.POINTS_PER_THREAD = ppt;
        }
    };

    #define runTest(blockSize, ppt) checkBestConf(blockSize, ppt, \
            runSelectIf< \
                    blockSize, \
                    ppt, \
                    DIM, \
                    LOAD_MODIFIER, \
                    IO_BACKEND, \
                    INPUT_MEM_LAYOUT, \
                    STORE_TWO_PHASE, T>( \
                numPoints, \
                compare, \
                d_in, \
                inStride, \
                outStride, \
                h_reference, \
                h_referenceSelectedPointsCnt));
    
    #ifdef QUICK_TEST
        runTest(128, 4);
    #else
        runTest(64, 8);
        
        runTest(128, 1);
        runTest(128, 4);
    #endif
    #undef runTest

    if (g_logResults)
    {
        *g_logFile << "% best: ";
         bestKernelParams.printLaunchConf(*g_logFile);
    }

    std::cout << "-------------------------------\n";
    std::cout << ">>>>>>> best performance conf: ";
    bestKernelParams.printLaunchConf(std::cout);
    std::cout << "\n-------------------------------\n";

    return bestKernelParams;
}


//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------

/**
 * Initialize problem
 */
template <typename    T>
static void Initialize(
    T*  h_in,
    int numItems)
{
    rd::fillRandomDataTable(h_in, numItems, T(0), T(126));
    if (g_verbose)
    {
        rd::printTable(h_in, numItems, "Input:");
    }
}

/**
 * @brief      Create if necessary and open log file. Allocate log file stream.
 */
template <typename T>
static void initializeLogFile()
{
    if (g_logResults)
    {
        std::ostringstream logFileName;
        // append device name to log file
        logFileName << g_devName << "_" << LOG_FILE_NAME_SUFFIX;

        std::string logFilePath = rd::findPath("timings/", logFileName.str());
        g_logFile = new std::ofstream(logFilePath.c_str(), std::ios::out | std::ios::app);
        if (g_logFile->fail())
        {
            throw std::logic_error("Couldn't open file: " + logFileName.str());
        }

        *g_logFile << "%" << rd::HLINE << std::endl;
        *g_logFile << "% " << typeid(T).name() << std::endl;
        *g_logFile << "%" << rd::HLINE << std::endl;
    }
}

/**
 * Reference selection problem solution.
 */
template <
    int         DIM,
    typename    T,
    typename    SelectOpT>
static void solve(
    T const *           h_in,
    size_t              numPoints,
    T *                 h_selectedPoints,
    int &               h_selectedPointsCnt,
    SelectOpT           selectOp)
{
    #ifdef RD_USE_OPENMP
    #pragma omp parallel for num_threads(8), schedule(static)
    #endif
    for (size_t k = 0; k < numPoints; ++k)
    {
        T const * point = h_in + k * DIM;

        if (selectOp(point))
        {
            int offset = 0;
            #ifdef RD_USE_OPENMP
            #pragma omp atomic capture
            #endif
            offset = h_selectedPointsCnt++;

            for (int d = 0; d < DIM; ++d)
            {
                h_selectedPoints[offset * DIM + d] = point[d];
            }
        }
    }
}

/**
 * @brief      Prepare and run test. Allocate and initialize test input data.
 */
template <
    int                         DIM,
    hipcub::CacheLoadModifier      LOAD_MODIFIER,
    rd::gpu::BlockTileIOBackend IO_BACKEND,
    rd::DataMemoryLayout        INPUT_MEM_LAYOUT,
    bool                        STORE_TWO_PHASE,
    typename                    T>
KernelParametersConf prepareAndRunTest(
    int                             numPoints,
    float                           selectRatio)
{
    // allocate host arrays
    T * h_in            = new T[numPoints * DIM];
    T * h_reference     = new T[numPoints * DIM];
    int h_referenceSelectedPointsCnt = 0;

    // Initialize input
    Initialize(h_in, numPoints * DIM);

    // Select a comparison value that is selectRatio through the space of [0,127]
    T compare;
    if (selectRatio <= 0.0)
    {
        compare = 0;        // select none
    }
    else if (selectRatio >= 1.0)
    {
        compare = 127;      // select all
    }
    else
    {
        compare = int(double(double(127) * selectRatio));
    }

    LessThan<DIM, T> selectOp(compare);
    solve<DIM>(h_in, numPoints, h_reference, h_referenceSelectedPointsCnt, selectOp);

    // sort results because, points selected on GPU may be stored in different order.
    std::sort(h_reference, h_reference + h_referenceSelectedPointsCnt * DIM);

    std::cout << "\nTest: \n"
          << ", load modifier: " << rd::LoadModifierNameTraits<LOAD_MODIFIER>::name
          << ", io backend: " << rd::BlockTileIONameTraits<IO_BACKEND>::name
          << ", mem layout: " << rd::DataMemoryLayoutNameTraits<INPUT_MEM_LAYOUT>::name
          << ", store two phase: " << std::boolalpha << STORE_TWO_PHASE
          << ", numPoints: " << numPoints << "("<<DIM<<"-dim)\n";
    std::cout << "\nComparison item: " << compare 
          << ", " << h_referenceSelectedPointsCnt << " selected points (select ratio " 
          << selectRatio << ")\n";

    if (g_logResults)
    {
        *g_logFile << "% "
              << " loadModifier=" << rd::LoadModifierNameTraits<LOAD_MODIFIER>::name
              << " ioBackend=" << rd::BlockTileIONameTraits<IO_BACKEND>::name
              << " memLayout=" << rd::DataMemoryLayoutNameTraits<INPUT_MEM_LAYOUT>::name
              << " storeTwoPhase=" << std::boolalpha << STORE_TWO_PHASE
              << " numPoints=" << numPoints 
              << " dim=" << DIM 
              << " compareItem=" << compare 
              << " selectRatio=" << selectRatio << "\n"; 
    }

    // Allocate device arrays
    T * d_in = nullptr;
    checkCudaErrors(hipMalloc(&d_in, numPoints * DIM * sizeof(T)));
    rd::gpu::rdMemcpy<DIM, INPUT_MEM_LAYOUT, rd::ROW_MAJOR, hipMemcpyHostToDevice>(
        d_in, h_in, numPoints);

    int inDataStride = (INPUT_MEM_LAYOUT == rd::COL_MAJOR) ? numPoints : 1;
    int outDataStride = (INPUT_MEM_LAYOUT == rd::COL_MAJOR) ? h_referenceSelectedPointsCnt : 1;

    // Run test kernel configurations
    KernelParametersConf bestKernelParams = testKernelConf<DIM, LOAD_MODIFIER, IO_BACKEND,
        INPUT_MEM_LAYOUT, STORE_TWO_PHASE>(
            numPoints,
            compare, 
            d_in, 
            inDataStride,
            outDataStride,
            h_reference,
            h_referenceSelectedPointsCnt);

    // cleanup

    if (h_in) delete[] h_in;
    if (h_reference) delete[] h_reference;
    if (d_in) checkCudaErrors(hipFree(d_in));

    return bestKernelParams;
}

template <
    int                             DIM,
    hipcub::CacheLoadModifier          LOAD_MODIFIER,
    rd::gpu::BlockTileIOBackend     IO_BACKEND,
    rd::DataMemoryLayout            INPUT_MEM_LAYOUT,
    typename                        T>    
void testStoreAlgorithm(
    int             numPoints,
    float           selectRatio)
{
    KernelParametersConf kp;

    /**
     * store two-phase (with selected items compaction in smem)
     */
    kp = prepareAndRunTest<DIM, LOAD_MODIFIER, IO_BACKEND, INPUT_MEM_LAYOUT, true, T>(
        numPoints, selectRatio);
    auto perfRes = std::make_tuple(selectRatio, DIM, kp);
    g_graphData.push_back(perfRes);

    /**
     * store using warp-aggregated algorithm
     */
    kp = prepareAndRunTest<DIM, LOAD_MODIFIER, IO_BACKEND, INPUT_MEM_LAYOUT, false, T>(
        numPoints, selectRatio);
    std::get<2>(perfRes) = kp;
    g_graphData.push_back(perfRes);
}

template <
    int                             DIM,
    rd::gpu::BlockTileIOBackend     IO_BACKEND,
    rd::DataMemoryLayout            INPUT_MEM_LAYOUT,
    typename                        T>
void testLoadModifier(
    int             numPoints,
    float           selectRatio)
{
    testStoreAlgorithm<DIM, hipcub::LOAD_LDG, IO_BACKEND, INPUT_MEM_LAYOUT, T>(numPoints, 
        selectRatio);
    // Cache streaming (likely to be accessed once)
    testStoreAlgorithm<DIM, hipcub::LOAD_CS, IO_BACKEND, INPUT_MEM_LAYOUT, T>(numPoints, 
        selectRatio);
}

template <
    int                             DIM,
    rd::DataMemoryLayout            INPUT_MEM_LAYOUT,
    typename                        T>
void testIOBackend(
    int             numPoints,
    float           selectRatio)
{
    testLoadModifier<DIM, rd::gpu::IO_BACKEND_CUB,   INPUT_MEM_LAYOUT, T>(numPoints, selectRatio);
    /**
     * 06.06.2016 Trove version causes misalinged address errors while storing data from smem.
     */
    // testLoadModifier<DIM, rd::gpu::IO_BACKEND_TROVE, INPUT_MEM_LAYOUT, T>(numPoints, selectRatio);
}

template <
    int         DIM,
    typename    T>
void testInputMemLayout(
    int             numPoints,
    float           selectRatio)
{
    testIOBackend<DIM, rd::ROW_MAJOR, T>(numPoints, selectRatio);
    testIOBackend<DIM, rd::COL_MAJOR, T>(numPoints, selectRatio);

}

template <
    typename       T>
void testDim(
    int             numPoints)
{
    if (g_logResults)
    {
        initializeLogFile<T>();
    }

    #if !defined(RD_DEBUG) && !defined(RD_PROFILE)
    const unsigned int MIN_TEST_DIM = 1;
    const unsigned int MAX_TEST_DIM = 6;
    #else
    const unsigned int MIN_TEST_DIM = 2;
    const unsigned int MAX_TEST_DIM = 2;
    #endif

    // for (float selectRatio = 0; selectRatio <= 1.0f; selectRatio += 0.2f)
    for (float selectRatio = 0.6; selectRatio <= 0.6f; selectRatio += 0.2f)
    {
        #if defined(RD_DEBUG) || defined(RD_PROFILE) 
            std::cout << "\nTest DIM 2...\n\n";
            testInputMemLayout<2, T>(numPoints, selectRatio);
        #else
            std::cout << "\nTest DIM 1...\n\n";
            testInputMemLayout<1, T>(numPoints, selectRatio);
            std::cout << "\nTest DIM 2...\n\n";
            testInputMemLayout<2, T>(numPoints, selectRatio);
            std::cout << "\nTest DIM 3...\n\n";
            testInputMemLayout<3, T>(numPoints, selectRatio);
            std::cout << "\nTest DIM 4...\n\n";
            testInputMemLayout<4, T>(numPoints, selectRatio);
            std::cout << "\nTest DIM 5...\n\n";
            testInputMemLayout<5, T>(numPoints, selectRatio);
            std::cout << "\nTest DIM 6...\n\n";
            testInputMemLayout<6, T>(numPoints, selectRatio);
        #endif

        //---------------------------------------------------
        //  summarize results
        //---------------------------------------------------
    
        for (unsigned int d = MIN_TEST_DIM; d <= MAX_TEST_DIM; ++d)
        {
            KernelParametersConf bestKernelParams;
            bestKernelParams.gigaBandwidth = -1.0f;

            // find best configuration for given dimension
            for (auto const & e : g_graphData)
            {
                if (std::get<1>(e) == d)
                {
                    KernelParametersConf conf = std::get<2>(e);
                    if (conf.gigaBandwidth > bestKernelParams.gigaBandwidth)
                    {
                        bestKernelParams = conf;
                    }
                }
            }

            if (g_logResults)
            {
                *g_logFile << "\n% overallBest "
                    << " dim=" << d 
                    << " avgMillis=" << bestKernelParams.avgMillis
                    << " gigaBandwidth=" << bestKernelParams.gigaBandwidth
                    << " blockThreads=" << bestKernelParams.BLOCK_THREADS
                    << " pointsPerThread=" << bestKernelParams.POINTS_PER_THREAD
                    << " loadModifier=" << rd::getLoadModifierName(bestKernelParams.LOAD_MODIFIER)
                    << " memLayout=" << rd::getRDDataMemoryLayout(bestKernelParams.INPUT_MEM_LAYOUT)
                    << " ioBackend=" << rd::getRDTileIOBackend(bestKernelParams.IO_BACKEND)
                    << " storeTwoPhase=" << std::boolalpha << bestKernelParams.STORE_TWO_PHASE
                    << " numPoints=" << numPoints << "\n";
            }

            std::cout << "\n>>>>> overall best conf: " 
                << "\n dim: \t\t\t" << d
                << "\n avgMillis: \t\t" << bestKernelParams.avgMillis
                << "\n gigaBandwidth: \t" << bestKernelParams.gigaBandwidth
                << "\n block threads: \t" << bestKernelParams.BLOCK_THREADS
                << "\n points per thread: \t" << bestKernelParams.POINTS_PER_THREAD
                << "\n load modifier: \t" << rd::getLoadModifierName(bestKernelParams.LOAD_MODIFIER)
                << "\n mem layout: \t\t" << rd::getRDDataMemoryLayout(bestKernelParams.INPUT_MEM_LAYOUT)
                << "\n io backend: \t\t" << rd::getRDTileIOBackend(bestKernelParams.IO_BACKEND)
                << "\n storeTwoPhase \t\t" << std::boolalpha << bestKernelParams.STORE_TWO_PHASE
                << "\n numPoints: \t\t" << numPoints << "\n";
        }

        if (g_drawGraphs)
        {
           createFinalGraphDataFile<T>(MIN_TEST_DIM, MAX_TEST_DIM, numPoints);
        }

        g_graphData.clear();
    }


    if (g_logResults)
    {
        if (g_logFile) delete g_logFile;
    }
}

template <
    typename        T>
void testSize(
    int             numPoints)
{
    if (numPoints < 0)
    {
        testDim<T>(0      );
        testDim<T>(1      );
        testDim<T>(100    );
        testDim<T>(10000  );
        testDim<T>(100000 );
        testDim<T>(1000000);
    }
    else
    {
        testDim<T>(numPoints);
    }
}

int main(int argc, char const **argv)
{
    
    int numPoints       = -1;
    int devId           = 0;
    float selectRatio   = 0.5f;

    // Initialize command line
    rd::CommandLineArgs args(argc, argv);
    if (args.CheckCmdLineFlag("help")) 
    {
        printf("%s \n"
            "\t\t--np=<number of input points>\n"
            "\t\t[--device=<device id>]\n"
            "\t\t[--ratio=<selection ratio, default 0.5>]\n"
            "\t\t[--v <verbose>]\n"
            "\t\t[--drawGraphs <draw graphs>]\n"
            "\t\t[--logResults <log performance results>]\n"
            "\n", argv[0]);
        exit(0);
    }

    args.GetCmdLineArgument("np", numPoints);

    if (args.CheckCmdLineFlag("device")) 
    {
        args.GetCmdLineArgument("device", devId);
    }
    if (args.CheckCmdLineFlag("ratio")) 
    {
        args.GetCmdLineArgument("ratio", selectRatio);
    }
    if (args.CheckCmdLineFlag("v")) 
    {
        g_verbose = true;
    }

    checkCudaErrors(deviceInit(devId));

    // set device name for logging and drawing purposes
    hipDeviceProp_t devProp;
    checkCudaErrors(hipGetDeviceProperties(&devProp, devId));
    g_devName = devProp.name;
    // read device SM count and determine max number of resident blocks per SM
    g_devSMCount = devProp.multiProcessorCount;
    g_devMaxBlocksPerSM = (devProp.major < 3) ? 8 : 16;

    //-----------------------------------------
    //  TESTS
    //-----------------------------------------

#ifdef QUICK_TEST

    if (numPoints < 0)
    {
        numPoints = 1000000;
    }

    std::cout << rd::HLINE << std::endl;
    std::cout << "FLOAT 2D: " << std::endl;
    prepareAndRunTest<2, hipcub::LOAD_LDG, rd::gpu::IO_BACKEND_CUB, rd::ROW_MAJOR, true,
        float>(numPoints, selectRatio);
    prepareAndRunTest<2, hipcub::LOAD_LDG, rd::gpu::IO_BACKEND_CUB, rd::COL_MAJOR, true, 
        float>(numPoints, selectRatio);
    prepareAndRunTest<2, hipcub::LOAD_LDG, rd::gpu::IO_BACKEND_CUB, rd::ROW_MAJOR, false,
        float>(numPoints, selectRatio);
    prepareAndRunTest<2, hipcub::LOAD_LDG, rd::gpu::IO_BACKEND_CUB, rd::COL_MAJOR, false, 
        float>(numPoints, selectRatio);

    /**
     * 06.06.2016 Trove version causes misalinged address errors while storing data from smem.
     */

    // prepareAndRunTest<2, hipcub::LOAD_LDG, rd::gpu::IO_BACKEND_TROVE, rd::ROW_MAJOR, true,
    //     float>(numPoints, selectRatio);
    // prepareAndRunTest<2, hipcub::LOAD_LDG, rd::gpu::IO_BACKEND_TROVE, rd::COL_MAJOR, true, 
    //     float>(numPoints, selectRatio);
    // prepareAndRunTest<2, hipcub::LOAD_LDG, rd::gpu::IO_BACKEND_TROVE, rd::ROW_MAJOR, false,
    //     float>(numPoints, selectRatio);
    // prepareAndRunTest<2, hipcub::LOAD_LDG, rd::gpu::IO_BACKEND_TROVE, rd::COL_MAJOR, false, 
    //     float>(numPoints, selectRatio);
    std::cout << rd::HLINE << std::endl;
    
#else    

    if (args.CheckCmdLineFlag("drawGraphs")) 
    {
        g_drawGraphs = true;
    }
    if (args.CheckCmdLineFlag("logResults")) 
    {
        g_logResults = true;
    }

    std::cout << rd::HLINE << std::endl;
    std::cout << "FLOAT: " << std::endl;
    testSize<float>(numPoints);
    std::cout << rd::HLINE << std::endl;
    std::cout << "DOUBLE: " << std::endl;
    testSize<double>(numPoints);
    std::cout << rd::HLINE << std::endl;

#endif

    checkCudaErrors(deviceReset());

    std::cout << "END!" << std::endl;
    return 0;
}
