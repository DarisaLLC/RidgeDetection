#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

__host__ __device__ __forceinline__ hipError_t Debug(
    hipError_t     error,
    const char*     filename,
    int             line)
{
    if (error)
    {
    #if (__CUDA_ARCH__ == 0)
        fprintf(stderr, "CUDA error %d [%s, %d]: %s\n",
            error, filename, line, hipGetErrorString(error));
        fflush(stderr);
    #elif (__CUDA_ARCH__ >= 200)
        printf("CUDA error %d [block (%3d,%3d,%3d) thread (%3d,%3d,%3d), %s, %d]\n",
            error, blockIdx.z, blockIdx.y, blockIdx.x,
            threadIdx.z, threadIdx.y, threadIdx.x, filename, line);
    #endif
    }
    return error;
}

#ifndef devCheckCall
    #define devCheckCall(e) if ( Debug((e), __FILE__, __LINE__) ) { assert(0); }
#endif

#ifndef checkCudaErrors
    #define checkCudaErrors(e) if ( Debug((e), __FILE__, __LINE__) ) { hipDeviceReset(); exit(1); }
#endif

//----------------------------------------------------------
//
//----------------------------------------------------------


__global__ void testKernel1(char *  in1, char * in2, char * in3, int size, int value)
{
    if (threadIdx.x == 0)
    {
        hipStream_t s1;
        devCheckCall(hipStreamCreateWithFlags(&s1, hipStreamNonBlocking));
        devCheckCall(hipMemsetAsync(in1, value, size * sizeof(char), s1))
        devCheckCall(hipStreamDestroy(s1));
    }

    if (threadIdx.x == 1)
    {
        hipStream_t s2;
        devCheckCall(hipStreamCreateWithFlags(&s2, hipStreamNonBlocking));
        devCheckCall(hipMemsetAsync(in2, value, size * sizeof(char), s2))
        devCheckCall(hipStreamDestroy(s2));
    }

    if (threadIdx.x == 2)
    {
        hipStream_t s3;
        devCheckCall(hipStreamCreateWithFlags(&s3, hipStreamNonBlocking));
        devCheckCall(hipMemsetAsync(in3, value, size * sizeof(char), s3))
        devCheckCall(hipStreamDestroy(s3));
    }
    __syncthreads();
    devCheckCall(hipDeviceSynchronize());
    
}


//----------------------------------------------------------
//
//----------------------------------------------------------


static constexpr int POINTS_NUM     = 1 << 16;

//----------------------------------------------------------
//
//----------------------------------------------------------


int main(void)
{

    int value = 1;

    char *h_data = new char[POINTS_NUM];
    char *d_out1 = new char[POINTS_NUM];
    char *d_out2 = new char[POINTS_NUM];
    char *d_out3 = new char[POINTS_NUM];
    char *d_in1, *d_in2, *d_in3;

    checkCudaErrors(hipSetDevice(0));

    checkCudaErrors(hipMalloc(&d_in1, POINTS_NUM * sizeof(char)));
    checkCudaErrors(hipMalloc(&d_in2, POINTS_NUM * sizeof(char)));
    checkCudaErrors(hipMalloc(&d_in3, POINTS_NUM * sizeof(char)));

    for (int k = 0; k < POINTS_NUM; ++k)
    {
        // h_data[k] = k * 0.2f;
        h_data[k] = char(k & 127);
    }

    std::cout << "char(200): " << char(200) << std::endl;
    std::cout << "sizeof(char): " << sizeof(char) << std::endl;

    checkCudaErrors(hipMemcpy(d_in1, h_data, POINTS_NUM * sizeof(char),
        hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_in2, h_data, POINTS_NUM * sizeof(char),
        hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_in3, h_data, POINTS_NUM * sizeof(char),
        hipMemcpyHostToDevice));
    checkCudaErrors(hipDeviceSynchronize());

    testKernel1<<<1, 32>>>(d_in1, d_in2, d_in3, POINTS_NUM, value);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(d_out1, d_in1, POINTS_NUM * sizeof(char),
        hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(d_out2, d_in2, POINTS_NUM * sizeof(char),
        hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(d_out3, d_in3, POINTS_NUM * sizeof(char),
        hipMemcpyDeviceToHost));

    for (int k = 0; k < POINTS_NUM; ++k)
    {
        if (int(d_out1[k]) != value ||
            int(d_out2[k]) != value ||
            int(d_out3[k]) != value)
        {
            std::cout << "--- ERROR!---- d_out1["<<k<<"]: (" << int(d_out1[k])
                << "), d_out2["<<k<<"]: (" << int(d_out2[k])
                << "), d_out3["<<k<<"]: (" << int(d_out3[k]) << ")" << std::endl;
            break;
        }
    }

    checkCudaErrors(hipDeviceReset());

    return 0;
}